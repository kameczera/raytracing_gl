#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>

#include "render.cuh"

GLuint pbo = 0;
GLuint tex = 0;
struct hipGraphicsResource* cuda_pbo_resource;
static int HEIGHT = 225;
static int WIDTH = 400;


void createPBO() {
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, WIDTH * HEIGHT * 4, NULL, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

void createTexture() {
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, WIDTH, HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
}

int main(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("CUDA + OpenGL Gradient");

    glewInit();
    createPBO();
    createTexture();

    glutDisplayFunc(display);
    glutMainLoop();

    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &tex);

    return 0;
}
